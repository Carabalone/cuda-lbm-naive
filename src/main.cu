#include "utility.cuh"

int main() {
    checkCudaErrors(hipSetDevice(0));
    int deviceId = 0;
    checkCudaErrors(hipGetDevice(&deviceId));

    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, deviceId));
    size_t gpu_free_mem, gpu_total_mem;
    checkCudaErrors(hipMemGetInfo(&gpu_free_mem, &gpu_total_mem));
    printf("CUDA information\n");
    printf("       using device: %d\n", deviceId);
    printf("               name: %s\n",deviceProp.name);
    printf("    multiprocessors: %d\n", deviceProp.multiProcessorCount);
    printf(" compute capability: %d.%d\n", deviceProp.major,deviceProp.minor);
    printf("      global memory: %.1f MiB\n", deviceProp.totalGlobalMem/1048576.0);
    printf("        free memory: %.1f MiB\n", gpu_free_mem/1048576.0);
    printf("\n");

    return 0;
}
